#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include <ATen/cuda/HIPContext.h>
#include <torch/torch.h>
#include <torch/extension.h>

#include <algorithm>
#include <stdexcept>

#include <stdint.h>
#include <cstdio>


#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be a contiguous tensor")
#define CHECK_IS_INT(x) TORCH_CHECK(x.scalar_type() == at::ScalarType::Int, #x " must be an int tensor")
#define CHECK_IS_FLOATING(x) TORCH_CHECK(x.scalar_type() == at::ScalarType::Float || x.scalar_type() == at::ScalarType::Half || x.scalar_type() == at::ScalarType::Double, #x " must be a floating tensor")


// requires CUDA >= 10 and ARCH >= 70
// this is very slow compared to float or __half2, do not use!
static inline  __device__ at::Half atomicAdd(at::Half *address, at::Half val) {
  return atomicAdd(reinterpret_cast<__half*>(address), val);
}


template <typename T>
static inline __host__ __device__ T div_round_up(T val, T divisor) {
	return (val + divisor - 1) / divisor;
}


template <typename scalar_t>
__global__ void kernel_grid_backward(
    const scalar_t * __restrict__ grad_outputs, // NxCxOHxOW
    const scalar_t * __restrict__ features,     // NxCxIHxIW
    const scalar_t * __restrict__ grid,         // NxOHxOWx2
    scalar_t * __restrict__ dy_dx,              // NxCxOHxOWx2
    scalar_t * __restrict__ grad_features,      // NxCxIHxIW
    scalar_t * __restrict__ grad_grid,          // NxOHxOWx2
    const uint32_t N, const uint32_t C, 
    const uint32_t IH, const uint32_t IW, 
    const uint32_t OH, const uint32_t OW) {
    const uint32_t b = blockIdx.x * blockDim.x + threadIdx.x;
	if (b>= N*OH*OW) return;

    // obtain index of the thread
    const uint32_t n = b / (OH * OW);
    const uint32_t oh = (b / OW) % OH;
    const uint32_t ow = b % OW;

    // skip to the corresponding grids
    grad_outputs = grad_outputs + n * C * OH * OW + oh * OW + ow;
    grid = grid + n * OH * OW * 2 + oh* OW * 2  + ow * 2;
    grad_grid = grad_grid + n * OH * OW * 2 + oh* OW * 2  + ow * 2;

    const scalar_t gx = (grid[0] + 1) / 2.0 * (IW - 1);
    const scalar_t gy = (grid[1] + 1) / 2.0 * (IH - 1);

    const uint32_t x0 = max(min((uint32_t)floor(gx), IW - 2), 0);
    const uint32_t y0 = max(min((uint32_t)floor(gy), IH - 2), 0);
    const uint32_t x1 = x0 + 1;
    const uint32_t y1 = y0 + 1;
    const scalar_t wx = min(max((gx - (scalar_t)x0), (scalar_t)0), (scalar_t)1);
    const scalar_t wy = min(max((gy - (scalar_t)y0), (scalar_t)0), (scalar_t)1);

    scalar_t sx = (scalar_t)(IW - 1) / 2.0;
    scalar_t sy = (scalar_t)(IH - 1) / 2.0;


    for(uint32_t c = 0; c < C; c++){
        const uint32_t o00  = n * C * IH * IW + c * IH * IW + y0 * IW + x0;
        const uint32_t o01  = n * C * IH * IW + c * IH * IW + y0 * IW + x1;
        const uint32_t o10  = n * C * IH * IW + c * IH * IW + y1 * IW + x0;
        const uint32_t o11  = n * C * IH * IW + c * IH * IW + y1 * IW + x1;
        const scalar_t go = (grad_outputs + c * OH * OW)[0];

        // compute grad features
        atomicAdd(grad_features + o00, go * (1 - wx) * (1 - wy));
        atomicAdd(grad_features + o01, go * (wx) * (1 - wy));
        atomicAdd(grad_features + o10, go * (1 - wx) * (wy));
        atomicAdd(grad_features + o11, go * wx * wy);

        // compute dy_dx
        const scalar_t f00 = features[o00];
        const scalar_t f01 = features[o01];
        const scalar_t f10 = features[o10];
        const scalar_t f11 = features[o11];

        const uint32_t g0 = n * C * OH * OW * 2 + c * OH * OW * 2 + oh * OW * 2 + ow * 2;
        const scalar_t g0x = -f00 * (1 - wy) + f01 * (1 - wy) - f10 * (wy)     + f11 * wy;
        const scalar_t g0y = -f00 * (1 - wx) - f01 * wx       + f10 * (1 - wx) + f11 * wx;
        const scalar_t dgx = g0x * sx;
        const scalar_t dgy = g0y * sy;
        dy_dx[g0] = dgx;
        dy_dx[g0 + 1] = dgy;

        // compute grad features
        atomicAdd(grad_features + o00, go * (1 - wx) * (1 - wy));
        atomicAdd(grad_features + o01, go * (wx) * (1 - wy));
        atomicAdd(grad_features + o10, go * (1 - wx) * (wy));
        atomicAdd(grad_features + o11, go * wx * wy);

        // compute grad grid
        grad_grid[0] += dgx * go;
        grad_grid[1] += dgy * go;
    }
}

template <typename scalar_t>
void grid_backward_cuda(
    const scalar_t *grad_outputs, 
    const scalar_t *features, 
    const scalar_t *grid, 
    scalar_t * dy_dx, 
    scalar_t * grad_features, 
    scalar_t * grad_grid, 
    const uint32_t N, const uint32_t C, 
    const uint32_t IH, const uint32_t IW, 
    const uint32_t OH, const uint32_t OW) {

    static constexpr uint32_t N_THREAD = 256;
	const dim3 blocks_hashgrid = { div_round_up(N * OH * OW, N_THREAD), 1, 1 };
    kernel_grid_backward<scalar_t><<<blocks_hashgrid, N_THREAD>>>(
        grad_outputs,
        features,
        grid,
        dy_dx,
        grad_features,
        grad_grid,
        N, C, IH, IW, OH, OW
    ); 
}

void grid_backward(
    const at::Tensor grad_outputs, 
    const at::Tensor features, 
    const at::Tensor grid, 
    at::Tensor dy_dx, 
    at::Tensor grad_features, 
    at::Tensor grad_grid, 
    const uint32_t N, const uint32_t C, 
    const uint32_t IH, const uint32_t IW, 
    const uint32_t OH, const uint32_t OW
){

    CHECK_CUDA(grad_outputs);
    CHECK_CUDA(features);
    CHECK_CUDA(grid);
    CHECK_CUDA(dy_dx);
    CHECK_CUDA(grad_features);
    CHECK_CUDA(grad_grid);

    CHECK_CONTIGUOUS(grad_outputs);
    CHECK_CONTIGUOUS(features);
    CHECK_CONTIGUOUS(grid);
    CHECK_CONTIGUOUS(dy_dx);
    CHECK_CONTIGUOUS(grad_features);
    CHECK_CONTIGUOUS(grad_grid);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        grad_outputs.scalar_type(), "grid_backward", (
            [&] {
                grid_backward_cuda<scalar_t>(
                    grad_outputs.data_ptr<scalar_t>(), 
                    features.data_ptr<scalar_t>(), 
                    grid.data_ptr<scalar_t>(), 
                    dy_dx.data_ptr<scalar_t>(), 
                    grad_features.data_ptr<scalar_t>(), 
                    grad_grid.data_ptr<scalar_t>(), 
                    N, C, IH, IW, OH, OW
                );
            }
        )
    );
}

// backward backward

template <typename scalar_t>
__global__ void kernel_grid_backward_backward(
    const scalar_t * __restrict__ grad_outputs,   // NxCxOHxOW
    const scalar_t * __restrict__ grad_grad_grid, // NxOHxOWx2
    const scalar_t * __restrict__ grid,           // NxOHxOWx2
    scalar_t * __restrict__ grad2_features,       // NxCxIHxIW
    const uint32_t N, const uint32_t C, 
    const uint32_t IH, const uint32_t IW, 
    const uint32_t OH, const uint32_t OW) {
    const uint32_t b = blockIdx.x * blockDim.x + threadIdx.x;
	if (b>= N*OH*OW) return;

    // obtain index of the thread
    const uint32_t n = b / (OH * OW);
    const uint32_t oh = (b / OW) % OH;
    const uint32_t ow = b % OW;

    // skip to the corresponding grids
    grad_outputs = grad_outputs + n * C * OH * OW + oh * OW + ow;
    grid = grid + n * OH * OW * 2 + oh* OW * 2  + ow * 2;
    grad_grad_grid = grad_grad_grid + n * OH * OW * 2 + oh* OW * 2  + ow * 2;

    const scalar_t gx = (grid[0] + 1) / 2.0 * (IW - 1);
    const scalar_t gy = (grid[1] + 1) / 2.0 * (IH - 1);

    const uint32_t x0 = max(min((uint32_t)floor(gx), IW - 2), 0);
    const uint32_t y0 = max(min((uint32_t)floor(gy), IH - 2), 0);
    const uint32_t x1 = x0 + 1;
    const uint32_t y1 = y0 + 1;
    const scalar_t wx = min(max((gx - (scalar_t)x0), (scalar_t)0), (scalar_t)1);
    const scalar_t wy = min(max((gy - (scalar_t)y0), (scalar_t)0), (scalar_t)1);

    const scalar_t ggx = grad_grad_grid[0];
    const scalar_t ggy = grad_grad_grid[1];
    const scalar_t sx = (scalar_t)(IW - 1) / 2.0 * ggx;
    const scalar_t sy = (scalar_t)(IH - 1) / 2.0 * ggy;

    for(uint32_t c = 0; c < C; c++){
        const uint32_t o00  = n * C * IH * IW + c * IH * IW + y0 * IW + x0;
        const uint32_t o01  = n * C * IH * IW + c * IH * IW + y0 * IW + x1;
        const uint32_t o10  = n * C * IH * IW + c * IH * IW + y1 * IW + x0;
        const uint32_t o11  = n * C * IH * IW + c * IH * IW + y1 * IW + x1;
        const scalar_t go = (grad_outputs + c * OH * OW)[0];

        atomicAdd(grad2_features + o00, go * (-(1 - wy) * sx - (1 - wx) * sy));
        atomicAdd(grad2_features + o01, go * ((1 - wy) * sx - wx * sy));
        atomicAdd(grad2_features + o10, go * (-wy * sx + (1 - wx) * sy));
        atomicAdd(grad2_features + o11, go * (wy * sx + wx * sy));
    }
}

template <typename scalar_t>
void grid_backward_backward_cuda(
    const scalar_t *grad_outputs, 
    const scalar_t *grad_grad_grids, 
    const scalar_t *grid, 
    scalar_t * grad2_feats, 
    const uint32_t N, const uint32_t C, 
    const uint32_t IH, const uint32_t IW, 
    const uint32_t OH, const uint32_t OW) {

    static constexpr uint32_t N_THREAD = 256;
	const dim3 blocks_hashgrid = { div_round_up(N * OH * OW, N_THREAD), 1, 1 };
    kernel_grid_backward_backward<scalar_t><<<blocks_hashgrid, N_THREAD>>>(
        grad_outputs,
        grad_grad_grids,
        grid,
        grad2_feats,
        N, C, IH, IW, OH, OW
    ); 
}

void grid_backward_backward(
    const at::Tensor grad_outputs, 
    const at::Tensor grad_grad_grid, 
    const at::Tensor grid, 
    at::Tensor grad2_features, 
    const uint32_t N, const uint32_t C, 
    const uint32_t IH, const uint32_t IW, 
    const uint32_t OH, const uint32_t OW
){

    CHECK_CUDA(grad_outputs);
    CHECK_CUDA(grad_grad_grid);
    CHECK_CUDA(grid);
    CHECK_CUDA(grad2_features);

    CHECK_CONTIGUOUS(grad_outputs);
    CHECK_CONTIGUOUS(grad_grad_grid);
    CHECK_CONTIGUOUS(grid);
    CHECK_CONTIGUOUS(grad2_features);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        grad_grad_grid.scalar_type(), "grid_backward_backward", (
            [&] {
                grid_backward_backward_cuda<scalar_t>(
                    grad_outputs.data_ptr<scalar_t>(), 
                    grad_grad_grid.data_ptr<scalar_t>(), 
                    grid.data_ptr<scalar_t>(), 
                    grad2_features.data_ptr<scalar_t>(), 
                    N, C, IH, IW, OH, OW
                );
            }
        )
    );
}
